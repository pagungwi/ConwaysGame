#include "hip/hip_runtime.h"
#include "./kernel.h" 

#define BLOCK 16
#define TILE_WIDTH (BLOCK+1)
#define RADIUS 1

/* Don't need global or shared create image:
   1. Library functions already exist
   2. Parallelization would need to initialize size of file, then each thread writes a byte of data at a different location
   3. Access to the file (read/write) would have race conditions and therefore be serialized anyways 
*/

__global__ 
void global_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){

}


__global__ 
void shared_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){

}


__global__ 
void global_Conways(uchar4*d_in, uchar4*d_out, const int rows, const int cols, const int phases){
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y; 

        int position = y*cols+x;
      
        
        
        int count = 0;
        
        // Neighbor positions
        int top_l = position - 1 - cols;
        int top_m = position - cols;
        int top_r = position + 1 - cols;
        int mid_l = position - 1;
        int mid_r = position + 1;
        int bot_l = position - 1 + cols;
        int bot_m = position + cols;
        int bot_r = position + cols + 1;
        
        if(x < cols && y < rows)
        {
                // Check if neighbors alive or dead, add to alive count
                if(x % cols != 0) // Need to check if on left wall of image
                {
                        if(top_l < rows*cols && top_l > -1 && d_in[top_l].x == 0) { count++; } // top left neighbor
                        if(mid_l < rows*cols && mid_l > -1 && d_in[mid_l].x == 0) { count++; } // mid left neigbor
                        if(bot_l < rows*cols && bot_l > -1 && d_in[bot_l].x == 0) { count++; } // bottom left neighbor
                }
                if(x % cols != cols-1) // Need to check if on right wall of image
                {   
                        if(top_r < rows*cols && top_r > -1 && d_in[top_r].x == 0) { count++; } // top right neighbor
                        if(mid_r < rows*cols && mid_r > -1 && d_in[mid_r].x == 0) { count++; } // middle right neighbor
                        if(bot_r < rows*cols && bot_r > -1 && d_in[bot_r].x == 0) { count++; }// bottom right neighbor
                }
                if(top_m < rows*cols && top_m > -1 && d_in[top_m].x == 0) { count++; } // top middle neighbor
                if(bot_m < rows*cols && bot_m > -1 && d_in[bot_m].x == 0) { count++; } // bottom middle neighbor

                // Update temp board based on counts
                if(d_in[position].x == 0 && (count < 2 || count > 3)) {d_out[position] = make_uchar4(255,255,255,255);} // if alive, make dead
                else if(d_in[position].x == 0 && (count == 2 || count == 3)) {d_out[position] = make_uchar4(0,0,0,255);} // if alive, stay alive
                else if(d_in[position].x == 255 && (count == 3)) {d_out[position] = make_uchar4(0,0,0,255);} // if dead, make alive
                else {d_out[position] = make_uchar4(255,255,255,255);} // if dead, stay dead

                //d_in[position] = d_temp[position];
        }
        /*if(x < cols && y < rows)
        {
                d_out[position] = d_in[position];
        }
        */
}

__global__ 
void shared_Conways(uchar4*d_in, uchar4*d_out, const int rows, const int cols, const int phases){
        // create shared variable to load halo of pixels around block for count calculations
        __shared__ uchar4 ds_in[TILE_WIDTH][TILE_WIDTH];
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int x = tx + blockIdx.x * blockDim.x;
        int y = ty + blockIdx.y * blockDim.y;

        int position = y*cols+x;
        int count = 0;
        
        /* Load original block dims into center of shared tile*/
        if(x < cols && y < rows) {
                ds_in[ty + RADIUS][tx + RADIUS] = d_in[y*cols+x];
        } else { 
                ds_in[ty + RADIUS][tx + RADIUS] = make_uchar4(255, 255, 255, 255); // Pad with dead cells
        }

        // Left halo (tx < 1)
        if(tx == 0) {
                if(x-RADIUS >= 0 && y < rows) {
                        ds_in[ty+RADIUS][tx] = d_in[y*cols+(x-RADIUS)];
                } else {
                        ds_in[ty+RADIUS][tx] = make_uchar4(255, 255, 255, 255);
                }
        }
        // Right halo (tx = cols)
        if(tx == cols) {
                if(x+RADIUS < cols && y < rows) {
                        ds_in[ty+RADIUS][tx + 2*RADIUS] = d_in[y*cols+(x+RADIUS)];
                } else {
                        ds_in[ty+RADIUS][tx + 2*RADIUS] = make_uchar4(255, 255, 255, 255);
                }
        }
        // Top halo (ty < 1)
        if(ty == 0) {
                if(x < cols && y-RADIUS >= 0) {
                        ds_in[ty][tx+RADIUS] = d_in[(y-RADIUS)*cols+x];
                } else {
                        ds_in[ty][tx+RADIUS] = make_uchar4(255, 255, 255, 255);
                }
        }
        // Bottom halo (ty = rows)
        if(ty == rows) {
                if(x < cols && y+RADIUS < rows) {
                        ds_in[ty + 2*RADIUS][tx+RADIUS] = d_in[(y+RADIUS)*cols+x];
                } else {
                        ds_in[ty + 2*RADIUS][tx+RADIUS] = make_uchar4(255, 255, 255, 255);
                }
        }
        // Top-left corner
        if(tx == 0 && ty == 0)
        {
                if(x-RADIUS >= 0 && y-RADIUS >=0) {
                        ds_in[ty][tx] = d_in[(y-RADIUS)*cols+(x-RADIUS)]; 
                } else {
                        ds_in[ty + 2*RADIUS][tx+RADIUS] = make_uchar4(255, 255, 255, 255);
                }
        }
        // Top-right corner
        if(tx == cols && ty == 0) {
                if(x+RADIUS < cols && y-RADIUS >= 0) {
                        ds_in[ty][tx] = d_in[(y-RADIUS)*cols+(x+RADIUS)];
                } else {
                        ds_in[ty + 2*RADIUS][tx+RADIUS] = make_uchar4(255, 255, 255, 255);
                }
        }
        // Bot-left corner
        if(tx == 0 && ty == rows) {
                if(x-RADIUS >= 0 && y+RADIUS < rows) {
                        ds_in[ty][tx] = d_in[(y+RADIUS)*cols+(x-RADIUS)];
                } else {
                        ds_in[ty + 2*RADIUS][tx+RADIUS] = make_uchar4(255, 255, 255, 255);
                }
        }
        // Bot-right corner
        if(tx == cols && ty == rows) {
                if(x+RADIUS < cols && y+RADIUS < rows) {
                        ds_in[ty][tx] = d_in[(y+RADIUS)*cols+(x+RADIUS)];
                } else {
                        ds_in[ty + 2*RADIUS][tx+RADIUS] = make_uchar4(255, 255, 255, 255);
                }
        }
        __syncthreads();
        
        uchar4 nextState;
        uchar4 initialState = ds_in[ty+1][tx+1];

        if(x < cols && y < rows)
        {
                // Check if neighbors alive or dead, add to alive count
                if(x % cols != 0) // Need to check if on left wall of image
                {
                        if(y % rows != 0 && ds_in[ty][tx].x == 0) { count++; } // top left neighbor
                        if(ds_in[ty+1][tx].x == 0) { count++; } // mid left neigbor
                        if(y % rows != rows-1 && ds_in[ty+2][tx].x == 0) { count++; } // bottom left neighbor
                }
                if(x % cols != cols-1) // Need to check if on right wall of image
                {   
                        if(y % rows != 0 && ds_in[ty][tx+2].x == 0) { count++; } // top right neighbor
                        if(ds_in[ty+1][tx+2].x == 0) { count++; } // middle right neighbor
                        if(y % rows != rows-1 && ds_in[ty+2][tx+2].x == 0) { count++; }// bottom right neighbor
                }
                if(y % rows != 0 && ds_in[ty][tx+1].x == 0) { count++; } // top middle neighbor
                if(y % rows != rows-1 && ds_in[ty+2][tx+1].x == 0) { count++; } // bottom middle neighbor

                // Update temp board based on counts
                if(initialState.x == 0 && (count < 2 || count > 3)) {nextState = make_uchar4(255,255,255,255);} // if alive, make dead
                else if(initialState.x == 0 && (count == 2 || count == 3)) {nextState = make_uchar4(0,0,0,255);} // if alive, stay alive
                else if(initialState.x == 255 && (count == 3)) {nextState = make_uchar4(0,0,0,255);} // if dead, make alive
                else {nextState = make_uchar4(255,255,255,255);} // if dead, stay dead
        }
        __syncthreads();
        if(x < cols && y < rows)
        {
                d_out[position] = nextState;
        }
}


void par_conway(uchar4 *d_in_img, uchar4 *d_out_img, size_t rows, size_t cols, size_t phases){
 
        // Defines number of blocks needed for data size
        dim3 gridSize((cols-1)/BLOCK+1, (rows-1)/BLOCK+1,1);
        // Defines number of threads per block
        dim3 blockSize(BLOCK, BLOCK, 1);

        for(int p = 0; p < phases; p++) {
                global_Conways<<<gridSize, blockSize>>>(d_in_img, d_out_img, rows, cols, phases);
                //shared_Conways<<<gridSize, blockSize>>>(d_in_img, d_out_img, rows, cols, phases);
                hipDeviceSynchronize();
                checkCudaErrors(hipGetLastError());
                
                 // Update gmaeboard state for next phase
                uchar4* temp = d_in_img;
                d_in_img = d_out_img;
                d_out_img = temp;
        }

/*
        gaussianBlur<<<gridSize, blockSize>>>(d_green, d_gblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_blue, d_bblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
*/
}




