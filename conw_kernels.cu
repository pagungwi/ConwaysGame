#include "hip/hip_runtime.h"
#include "./kernel.h"

#define BLOCK 8
#define TILE_WIDTH (BLOCK + 2*RADIUS)
#define RADIUS 1

/* Don't need global or shared create image:
   1. Library functions already exist
   2. Parallelization would need to initialize size of file, then each thread writes a byte of data at a different location
   3. Access to the file (read/write) would have race conditions and therefore be serialized anyways 
*/

__global__ 
void global_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){}

__global__ 
void shared_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){}

__global__ 
void global_Conways(uchar4*d_in, uchar4*d_out, const int rows, const int cols, const int phases){
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y; 

        int position = y*cols+x;
        int count = 0;
        
        // Neighbor positions
        int top_l = position - 1 - cols;
        int top_m = position - cols;
        int top_r = position + 1 - cols;
        int mid_l = position - 1;
        int mid_r = position + 1;
        int bot_l = position - 1 + cols;
        int bot_m = position + cols;
        int bot_r = position + cols + 1;
        
        if(x < cols && y < rows)
        {
                // Check if neighbors alive or dead, add to alive count
                if(x % cols != 0) // Need to check if on left wall of image
                {
                        if(top_l < rows*cols && top_l > -1 && d_in[top_l].x == 0) { count++; } // top left neighbor
                        if(mid_l < rows*cols && mid_l > -1 && d_in[mid_l].x == 0) { count++; } // mid left neigbor
                        if(bot_l < rows*cols && bot_l > -1 && d_in[bot_l].x == 0) { count++; } // bottom left neighbor
                }
                if(x % cols != cols-1) // Need to check if on right wall of image
                {   
                        if(top_r < rows*cols && top_r > -1 && d_in[top_r].x == 0) { count++; } // top right neighbor
                        if(mid_r < rows*cols && mid_r > -1 && d_in[mid_r].x == 0) { count++; } // middle right neighbor
                        if(bot_r < rows*cols && bot_r > -1 && d_in[bot_r].x == 0) { count++; }// bottom right neighbor
                }
                if(top_m < rows*cols && top_m > -1 && d_in[top_m].x == 0) { count++; } // top middle neighbor
                if(bot_m < rows*cols && bot_m > -1 && d_in[bot_m].x == 0) { count++; } // bottom middle neighbor

                // Update temp board based on counts
                if(d_in[position].x == 0 && (count < 2 || count > 3)) {d_out[position] = make_uchar4(255,255,255,255);} // if alive, make dead
                else if(d_in[position].x == 0 && (count == 2 || count == 3)) {d_out[position] = make_uchar4(0,0,0,255);} // if alive, stay alive
                else if(d_in[position].x == 255 && (count == 3)) {d_out[position] = make_uchar4(0,0,0,255);} // if dead, make alive
                else {d_out[position] = make_uchar4(255,255,255,255);} // if dead, stay dead
        }
}

__global__ 
void shared_Conways(uchar4*d_in, uchar4*d_out, const int rows, const int cols, const int phases){
        // create shared variable to load halo of pixels around block for count calculations
        __shared__ uchar4 ds_in[TILE_WIDTH][TILE_WIDTH];
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        int x = tx + blockIdx.x * blockDim.x;
        int y = ty + blockIdx.y * blockDim.y;

        int position = y*cols+x;
        int count = 0;
        // Step 1: Load the original image block into the center of ds_in
        if (y < rows && x < cols) {
                ds_in[ty + RADIUS][tx + RADIUS] = d_in[y * cols + x];
        } else {
                ds_in[ty + RADIUS][tx + RADIUS] = make_uchar4(255, 255, 255, 255);   // Zero-padding for out-of-boundary
        }

        // Left halo
        if (tx == 0) {
                int halo_x = x - RADIUS;
                if (halo_x >= 0 && y < rows) {
                ds_in[ty + RADIUS][tx] = d_in[y * cols + halo_x];
                } else {
                ds_in[ty + RADIUS][tx] = make_uchar4(255, 255, 255, 255);   // Zero-padding for out-of-boundary
                }
        }
        // Right halo
        if (tx == BLOCK - RADIUS) {
                int halo_x = x + RADIUS;
                if (halo_x < cols && y < rows) {
                ds_in[ty + RADIUS][tx + 2 * RADIUS] = d_in[y * cols + halo_x];
                } else {
                ds_in[ty + RADIUS][tx + 2 * RADIUS] = make_uchar4(255, 255, 255, 255);  // Zero-padding for out-of-boundary
                }
        }
        // Top halo
        if (ty == 0) {
                int halo_y = y - RADIUS;
                if (halo_y >= 0 && x < cols) {
                ds_in[ty][tx + RADIUS] = d_in[halo_y * cols + x];
                } else {
                ds_in[ty][tx + RADIUS] = make_uchar4(255, 255, 255, 255);   // Zero-padding for out-of-boundary
                }
        }
        // Bottom halo
        if (ty == BLOCK - RADIUS) {
                int halo_y = y + RADIUS;
                if (halo_y < rows && x < cols) {
                ds_in[ty + 2 * RADIUS][tx + RADIUS] = d_in[halo_y * cols + x];
                } else {
                ds_in[ty + 2 * RADIUS][tx + RADIUS] = make_uchar4(255, 255, 255, 255);   // Zero-padding for out-of-boundary
                }
        }
        // Top-left corner
        if (tx == 0 && ty == 0) {
                int halo_x = x - RADIUS;
                int halo_y = y - RADIUS;
                if (halo_x >= 0 && halo_y >= 0) {
                ds_in[ty][tx] = d_in[halo_y * cols + halo_x];
                } else {
                ds_in[ty][tx] = make_uchar4(255, 255, 255, 255);   // Zero-padding for out-of-boundary
                }
        }
        // Top-right corner
        if (tx == BLOCK - RADIUS && ty == 0) {
                int halo_x = x + RADIUS;
                int halo_y = y - RADIUS;
                if (halo_x < cols && halo_y >= 0) {
                ds_in[ty][tx + 2 * RADIUS] = d_in[halo_y * cols + halo_x];
                } else {
                ds_in[ty][tx + 2 * RADIUS] = make_uchar4(255, 255, 255, 255);   // Zero-padding for out-of-boundary
                }
        }
        // Bottom-left corner
        if (tx == 0 && ty == BLOCK - RADIUS) {
                int halo_x = x - RADIUS;
                int halo_y = y + RADIUS;
                if (halo_x >= 0 && halo_y < rows) {
                ds_in[ty + 2 * RADIUS][tx] = d_in[halo_y * cols + halo_x];
                } else {
                ds_in[ty + 2 * RADIUS][tx] = make_uchar4(255, 255, 255, 255);  // Zero-padding for out-of-boundary
                }
        }
        // Bottom-right corner
        if (tx == BLOCK - RADIUS && ty == BLOCK - RADIUS) {
                int halo_x = x + RADIUS;
                int halo_y = y + RADIUS;
                if (halo_x < cols && halo_y < rows) {
                ds_in[ty + 2 * RADIUS][tx + 2 * RADIUS] = d_in[halo_y * cols + halo_x];
                } else {
                ds_in[ty + 2 * RADIUS][tx + 2 * RADIUS] = make_uchar4(255, 255, 255, 255);  // Zero-padding for out-of-boundary
                }
        }
        __syncthreads();

        if(x < cols && y < rows)
        {
                int row_shared = ty + RADIUS;
                int col_shared = tx + RADIUS;
                
                if(ds_in[row_shared][col_shared-1].x == 0) { count++; } // mid left neigbor
                if(ds_in[row_shared-1][col_shared-1].x == 0) { count++; } // top left neighbor
                if(ds_in[row_shared+1][col_shared-1].x == 0) { count++; } // bottom left neighbor
                if(ds_in[row_shared][col_shared+1].x == 0) { count++; } // middle right neighbor
                if(ds_in[row_shared-1][col_shared+1].x == 0) { count++; } // top right neighbor
                if(ds_in[row_shared+1][col_shared+1].x == 0) { count++; }// bottom right neighbor
                if(ds_in[row_shared-1][col_shared].x == 0) { count++; } // top middle neighbor
                if(ds_in[row_shared+1][col_shared].x == 0) { count++; } // bottom middle neighbor

                if(ds_in[row_shared][col_shared].x == 0 && (count < 2 || count > 3)) {d_out[position] = make_uchar4(255,255,255,255);} // if alive, make dead
                else if(ds_in[row_shared][col_shared].x == 0 && (count == 2 || count == 3)) {d_out[position] = make_uchar4(0,0,0,255);} // if alive, stay alive
                else if(ds_in[row_shared][col_shared].x == 255 && (count == 3)) {d_out[position] = make_uchar4(0,0,0,255);} // if dead, make alive
                else {d_out[position] = make_uchar4(255,255,255,255);} // if dead, stay dead    
        }
}

void par_conway(uchar4 *d_in_img, uchar4 *d_out_img, size_t rows, size_t cols, size_t phases){
 
        // Defines number of blocks needed for data size
        dim3 gridSize((cols-1)/BLOCK+1, (rows-1)/BLOCK+1,1);
        // Defines number of threads per block
        dim3 blockSize(BLOCK, BLOCK, 1);

        std::chrono::time_point<std::chrono::system_clock> start, end;
        std::chrono::duration<double> elapsed_time;

        std::cout << "Parallel Kernel Execution Time:\n";

        start = std::chrono::system_clock::now();
        for(int p = 0; p < phases; p++) {
                //global_Conways<<<gridSize, blockSize>>>(d_in_img, d_out_img, rows, cols, phases);
                shared_Conways<<<gridSize, blockSize>>>(d_in_img, d_out_img, rows, cols, phases);
                hipDeviceSynchronize();
                checkCudaErrors(hipGetLastError());
                
                 // Update gmaeboard state for next phase
                uchar4* temp = d_in_img;
                d_in_img = d_out_img;
                d_out_img = temp;
        }
        end = std::chrono::system_clock::now();
        elapsed_time = end-start;
        std::cout << elapsed_time.count() << "s\n";
}




