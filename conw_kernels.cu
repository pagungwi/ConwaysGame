#include "hip/hip_runtime.h"
#include "./kernel.h" 

#define BLOCK 8
#define TILE_WIDTH BLOCK

/* Don't need global or shared create image:
   1. Library functions already exist
   2. Parallelization would need to initialize size of file, then each thread writes a byte of data at a different location
   3. Access to the file (read/write) would have race conditions and therefore be serialized anyways 
*/

__global__ 
void global_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){

}


__global__ 
void shared_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){

}


__global__ 
void global_Conways(uchar4*d_in, uchar4*d_out, uchar4*d_temp, const int rows, const int cols, const int phases){
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        int y = threadIdx.y + blockIdx.y * blockDim.y; 

        int position = y*cols+x;
      
        for(int p = 0; p < phases; p++) {
                
                int count = 0;
                
                // Neighbor positions
                int top_l = position - 1 - cols;
                int top_m = position - cols;
                int top_r = position + 1 - cols;
                int mid_l = position - 1;
                int mid_r = position + 1;
                int bot_l = position - 1 + cols;
                int bot_m = position + cols;
                int bot_r = position + cols + 1;
                
                if(position < rows*cols && position > -1)
                {
                        // Check if neighbors alive or dead, add to alive count
                        if(x % cols != 0) // Need to check if on left wall of image
                        {
                                if(top_l < rows*cols && top_l > -1 && d_in[top_l].x == 0) { count++; } // top left neighbor
                                if(mid_l < rows*cols && mid_l > -1 && d_in[mid_l].x == 0) { count++; } // mid left neigbor
                                if(bot_l < rows*cols && bot_l > -1 && d_in[bot_l].x == 0) { count++; } // bottom left neighbor
                        }
                        if(x % (cols-1) != 0) // Need to check if on right wall of image
                        {   
                                if(top_r < rows*cols && top_r > -1 && d_in[top_r].x == 0) { count++; } // top right neighbor
                                if(mid_r < rows*cols && mid_r > -1 && d_in[mid_r].x == 0) { count++; } // middle right neighbor
                                if(bot_r < rows*cols && bot_r > -1 && d_in[bot_r].x == 0) { count++; }// bottom right neighbor
                        }
                        if(top_m < rows*cols && top_m > -1 && d_in[top_m].x == 0) { count++; } // top middle neighbor
                        if(bot_m < rows*cols && bot_m > -1 && d_in[bot_m].x == 0) { count++; } // bottom middle neighbor

                        // Update temp board based on counts
                        if(d_in[position].x == 0 && (count < 2 || count > 3)) {d_temp[position] = make_uchar4(255,255,255,255);} // if alive, make dead
                        else if(d_in[position].x == 0 && (count == 2 || count == 3)) {d_temp[position] = make_uchar4(0,0,0,255);} // if alive, stay alive
                        else if(d_in[position].x == 255 && (count == 3)) {d_temp[position] = make_uchar4(0,0,0,255);} // if dead, make alive
                        else {d_temp[position] = make_uchar4(255,255,255,255);} // if dead, stay dead
                        d_in[position] = d_temp[position];
                }
                __syncthreads();
        }
        if(position < rows*cols && position > -1)
        {
                d_out[position] = d_in[position];
        }
}

__global__ 
void shared_Conways(uchar4*d_in, uchar4*d_out, const int rows, const int cols){

}


void par_conway(uchar4 *d_in_img, uchar4 *d_out_img, uchar4 *d_temp, size_t rows, size_t cols, size_t phases){
 
        // Defines number of blocks needed for data size
        dim3 gridSize((cols-1)/BLOCK+1, (rows-1)/BLOCK+1,1);
        // Defines number of threads per block
        dim3 blockSize(BLOCK, BLOCK, 1);


        global_Conways<<<gridSize, blockSize>>>(d_in_img, d_out_img, d_temp, rows, cols, phases);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

/*
        gaussianBlur<<<gridSize, blockSize>>>(d_green, d_gblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_blue, d_bblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
*/
}




