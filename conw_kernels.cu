#include "hip/hip_runtime.h"
#include "./kernel.h" 

#define BLOCK 4
#define TILE_WIDTH BLOCK

/*
The actual gaussian blur kernel to be implemented by 
you. Keep in mind that the kernel operates on a 
single channel.
 */


__global__ 
void global_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){

}


__global__ 
void shared_CreateImage(uchar4*d_in, uchar4*d_out, const int rows, const int cols, uchar4 * image){

}


__global__ 
void global_Conways(uchar4*d_in, uchar4*d_out, const int rows, const int cols){

}

__global__ 
void shared_Conways(uchar4*d_in, uchar4*d_out, const int rows, const int cols){

}


void par_conway(uchar4 * d_gameboard, uchar4 *d_in_img, uchar4 *d_out_img, size_t rows, size_t cols, size_t phases){
 
        // Defines number of blocks needed for data size
        dim3 gridSize((cols-1)/BLOCK+1, (rows-1)/BLOCK+1,1);
        // Defines number of threads per block
        dim3 blockSize(BLOCK, BLOCK, 1);

/*
        gaussianBlur<<<gridSize, blockSize>>>(d_red, d_rblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_green, d_gblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

        gaussianBlur<<<gridSize, blockSize>>>(d_blue, d_bblurred, rows, cols, d_filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
*/
}




